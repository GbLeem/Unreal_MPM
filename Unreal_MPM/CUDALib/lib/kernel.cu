#include "hip/hip_runtime.h"
﻿#include "cuda_lib_test.h"

__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void addKernel2(int4* c, const int4* a, const int4* b)
{
    int i = threadIdx.x;
    c[i].x = a[i].x + b[i].x;
    c[i].y = a[i].y + b[i].y;
    c[i].z = a[i].z + b[i].z;
    c[i].w = a[i].w + b[i].w;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size, std::string* error_message)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cuda_status;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cuda_status = hipSetDevice(0);
    if (cuda_status != hipSuccess) {
        *error_message = "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?";
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cuda_status = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMalloc failed!";
        goto Error;
    }

    cuda_status = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMalloc failed!";
        goto Error;
    }

    cuda_status = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMalloc failed!";
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cuda_status = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpy failed!";
        goto Error;
    }

    cuda_status = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpy failed!";
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel << <1, size >> > (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
        *error_message = "addKernel launch failed: " + std::string(hipGetErrorString(cuda_status));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cuda_status = hipDeviceSynchronize();
    if (cuda_status != hipSuccess) {
        *error_message = "hipDeviceSynchronize returned error code " + std::to_string(cuda_status) + " after launching addKernel!";
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cuda_status = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpy failed!";
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cuda_status;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda2(int4* c, const int4* a, const int4* b, std::string* error_message)
{
    int4* dev_a = 0;
    int4* dev_b = 0;
    int4* dev_c = 0;
    const unsigned int size = 1;
    hipError_t cuda_status;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cuda_status = hipSetDevice(0);
    if (cuda_status != hipSuccess) {
        *error_message = "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?";
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cuda_status = hipMalloc((void**)&dev_c, size * sizeof(int4));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMalloc failed!";
        goto Error;
    }

    cuda_status = hipMalloc((void**)&dev_a, size * sizeof(int4));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMalloc failed!";
        goto Error;
    }

    cuda_status = hipMalloc((void**)&dev_b, size * sizeof(int4));
    if (cuda_status != hipSuccess) {
        *error_message = "hipMalloc failed!";
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cuda_status = hipMemcpy(dev_a, a, size * sizeof(int4), hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpy failed!";
        goto Error;
    }

    cuda_status = hipMemcpy(dev_b, b, size * sizeof(int4), hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpy failed!";
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel2 << <1, size >> > (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
        *error_message = "addKernel launch failed: " + std::string(hipGetErrorString(cuda_status));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cuda_status = hipDeviceSynchronize();
    if (cuda_status != hipSuccess) {
        *error_message = "hipDeviceSynchronize returned error code " + std::to_string(cuda_status) + " after launching addKernel!";
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cuda_status = hipMemcpy(c, dev_c, size * sizeof(int4), hipMemcpyDeviceToHost);
    if (cuda_status != hipSuccess) {
        *error_message = "hipMemcpy failed!";
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cuda_status;
}
